
#include <hip/hip_runtime.h>
#include<stdio.h>

int main(){
    float* fpDevice;
    size_t nBytes = 4;
    hipError_t error = hipMalloc(&fpDevice, nBytes);
    hipMemset(fpDevice, 0, nBytes);
    
    float* fpHost;
    fpHost = (float*)malloc(nBytes);
    hipMemcpy(fpHost, fpDevice, nBytes, hipMemcpyDeviceToHost);

    printf("the number is %.2f", *fpHost);
    return 0;
}