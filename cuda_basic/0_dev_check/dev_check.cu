
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add_vector_gpu(float* A, float* B, float* C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i]+ B[i];

}

int main(){
    int deviceCnt = 0;
    hipError_t error = hipGetDeviceCount(&deviceCnt);
    if (error != hipSuccess || deviceCnt == 0){
        printf("found no compatible gpu!\n");
        exit(-1);
    }else{
        printf("found %d compatible gpu\n", deviceCnt);
    }
    int devId = 0;
    error = hipSetDevice(devId);
    if(error!=hipSuccess){
        printf("can't set device 0\n");
        exit(-1);
    }else {
        printf("successfully set cuda 0 as default device\n");
    }

    
    
}