#include<stdio.h>
#include"../5_error_check/common.cuh"

int main(){
    hipEvent_t start, stop;
    errorCheck(hipEventCreate(&start),__FILE__, __LINE__);
    errorCheck(hipEventCreate(&stop), __FILE__, __LINE__);
    errorCheck(hipEventRecord(start),__FILE__, __LINE__);
    hipEventQuery(start);

    // wast time code 

    errorCheck(hipEventRecord(stop),__FILE__,__LINE__);
    errorCheck(hipEventSynchronize(stop), __FILE__, __LINE__);

    float time_elapsed;
    errorCheck(hipEventElapsedTime(&time_elapsed, start, stop),__FILE__, __LINE__);
    printf("time elapsed %g ms.\n", time_elapsed);
    return 0;

}