
#include <hip/hip_runtime.h>
#include<stdio.h>

void initData(float* fp, int elementsNum){
    if (fp==NULL){
        printf("init data error, null pointer\n");
    }
    memset(fp,0, elementsNum*sizeof(float));
    for (int i=0; i< elementsNum; i++){
        fp[i] = (float)(rand() & 0xff) / 10.0f;
    }

}
__global__ void plusInGpu(float* A, float*B, float*C, int elementsNum){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i< elementsNum){
        C[i] = A[i] + B[i];
    }
}


int main(){

    // 1. device check
    int deviceCnt = 0;
    hipError_t error = hipGetDeviceCount(&deviceCnt);
    if (error!=hipSuccess || deviceCnt == 0){
        printf("can not find compatible device!\n");
        exit(-1);
    }else{
        printf("found device count: %d\n", deviceCnt);
    }

    int deviceId = 0;
    error = hipSetDevice(deviceId);
    if( error!= hipSuccess){
        printf("can not set device 0 for computation\n");
        exit(-1);
    }

    // 2. memory allocation
    float* fpHostA = NULL;
    float* fpHostB = NULL;
    float* fpHostC = NULL;

    int elementsNum = 2048;
    size_t nBytes = elementsNum * sizeof(float);
    fpHostA = (float*)malloc(nBytes);
    fpHostB = (float*)malloc(nBytes);
    fpHostC = (float*)malloc(nBytes);
    memset(fpHostA,0, nBytes);
    memset(fpHostB,0, nBytes);
    memset(fpHostC,0, nBytes);


    if(fpHostA == NULL || fpHostB == NULL || fpHostC == NULL){
        printf("failed to malloc host memory\n");
        exit(-1);
    }

    float* fpDeviceA =NULL;
    float* fpDeviceB =NULL;
    float* fpDeviceC =NULL;
    error = hipMalloc(&fpDeviceA, nBytes);
    if (error!= hipSuccess){
        printf("can not malloc size %d Bytes device memory for A\n", elementsNum);
    }
    error = hipMalloc(&fpDeviceB, nBytes);
    if (error!= hipSuccess){
        printf("can not malloc size %d Bytes device memory for B\n", elementsNum);
    }
    error = hipMalloc(&fpDeviceC, nBytes);
    if (error!= hipSuccess){
        printf("can not malloc size %d Bytes device memory for C\n", elementsNum);
    }
    hipMemset(fpDeviceA,0,nBytes);
    hipMemset(fpDeviceB,0,nBytes);
    hipMemset(fpDeviceC,0,nBytes);


    // 3. init data in host 
    srand(123);
    initData(fpHostA, elementsNum);
    initData(fpHostB, elementsNum);
    for (int i = 0; i < 10; i++){
        printf("%f\t",fpHostA[i]);
    }
    

    // 4. copy data to device
    error = hipMemcpy(fpDeviceA, fpHostA, nBytes, hipMemcpyHostToDevice);
    if (error!=hipSuccess){
        printf("failed to copy data from host to device");
        exit(-1);
    }
    error = hipMemcpy(fpDeviceB, fpHostB, nBytes, hipMemcpyHostToDevice);
    
    
    // 5. call kernel function
    dim3 block(256);
    dim3 grid((elementsNum + 256-1) /256);
    plusInGpu<<<grid, block>>>(fpDeviceA, fpDeviceB, fpDeviceC, elementsNum);
    hipDeviceSynchronize();

    // 6. copy data to host memory 
    hipMemcpy(fpHostC, fpDeviceC, nBytes, hipMemcpyDeviceToHost);

    for(int i=0;i< 10; i++){
        printf("c[%d] is %f\t",i, fpHostC[i]);
    }

    // 7. free memory
    free(fpHostA);
    free(fpHostB);
    free(fpHostC);

    hipFree(fpDeviceA);
    hipFree(fpDeviceB);
    hipFree(fpDeviceC);
    return 0;

}