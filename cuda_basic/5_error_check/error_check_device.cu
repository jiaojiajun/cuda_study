#include<stdio.h>
#include"./common.cuh"

int main(){
    float* fpHost;
    size_t nBytes = 1024 * sizeof(float);
    fpHost = (float*)malloc(nBytes);

    float* fpDevice;
    hipError_t error = hipMalloc(&fpDevice, nBytes);
    errorCheck(hipMemcpy(fpDevice, fpHost, nBytes, hipMemcpyDeviceToHost),__FILE__, __LINE__);
}