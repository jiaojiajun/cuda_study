
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<cassert>

const int N = 1 << 16;
const size_t nBytes = N * sizeof(int);
const int SIZE = 256;

void initVecotor(int* p){
    for (int i = 0; i < N; i++){
        p[i] = 1;
    }
}

__device__ void warpReduce(volatile int* sharedMem, int tid){
    sharedMem[tid] += sharedMem[tid+32];
    sharedMem[tid] += sharedMem[tid+16];
    sharedMem[tid] += sharedMem[tid+8];
    sharedMem[tid] += sharedMem[tid+4];
    sharedMem[tid] += sharedMem[tid+2];
    sharedMem[tid] += sharedMem[tid+1];
}

void verifyResult(int *p){
    printf("the res is %d\n", p[0]);
}

__global__ void sumReduce(int *p, int * res){
    const int MEM_SIZE = SIZE * 4; 
    __shared__ int sharedMem[MEM_SIZE];
    
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    sharedMem[threadIdx.x] = p[i] + p[i+blockDim.x];
    __syncthreads();
    for(int s = blockDim.x /2; s > 32; s>>=1){
        if(threadIdx.x < s){
            sharedMem[threadIdx.x ] += sharedMem[threadIdx.x+s];
        }
        __syncthreads();
    }
    if(threadIdx.x < 32){
        warpReduce(sharedMem,threadIdx.x);
    }
    if (threadIdx.x == 0){
        res[blockIdx.x] = sharedMem[0];
    }
}

int main(){
    int *ipHost,*ipHostRes, *ipDevice, *ipDeviceRes;
    ipHost = (int*) malloc(nBytes);
    ipHostRes = (int*) malloc(nBytes);

    hipMalloc(&ipDevice, nBytes);
    hipMalloc(&ipDeviceRes, nBytes);
    initVecotor(ipHost);
    hipMemcpy(ipDevice, ipHost, nBytes, hipMemcpyHostToDevice);

    int THREADS = SIZE;
    int BLOCKS = N / SIZE / 2;

    sumReduce<<<BLOCKS, THREADS>>>(ipDevice,ipDeviceRes); 
    sumReduce<<<1,BLOCKS>>>(ipDeviceRes, ipDeviceRes);
    
    hipMemcpy(ipHostRes, ipDeviceRes, sizeof(int), hipMemcpyDeviceToHost);

    verifyResult(ipHostRes);

    return 0;
}